//write a quick test for the parser
#include <iostream>
#include <cmath>
#include <vector>
#include <string>
#include <fstream>
#include <math.h>
#include <cmath>
#include "parser.hpp"
#include <chrono>
#include "cpufuncs.hpp"
// cuda includes
#include "cudafuncs.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

int main(int argc, char** argv) {
    // Check if the correct number of arguments is provided
    std::string configFile;
    if (argc != 2) {
         configFile= "config.txt";
    }
    else {    // Get the config file name from command line arguments
         configFile = argv[1];
    }
    // Test file name
    // Variables to hold parsed data
    std::vector<double> positions_old, velocities_old, masses, positions_new, velocities_new, accelerations, forces;
    
    double timeStepLength = 0.0, timeStepCount = 0.0, sigma = 0.0, epsilon = 0.0, boxSize = 0.0, cutoffRadius =0.0;
    int printInterval = 0;
    int numParticles = 0;
    int useAcc = 1;

    
    // Call the parser
    parseConfigFile(configFile, positions_old, velocities_old, masses, boxSize, cutoffRadius, timeStepLength, timeStepCount, sigma, epsilon, printInterval, useAcc);

    // Output the parsed data
    std::cout << "Parsed Data:" << std::endl;
    std::cout << "Time Step Length: " << timeStepLength << std::endl;
    std::cout << "Time Step Count: " << timeStepCount << std::endl;
    std::cout << "Sigma: " << sigma << std::endl;
    std::cout << "Epsilon: " << epsilon << std::endl;
    std::cout << "Box Size: " << boxSize << std::endl;
    std::cout<< "Cutoff Radius: "<< cutoffRadius<< std::endl;
    std::cout << "Print Interval: " << printInterval << std::endl;  
    numParticles = positions_old.size()/3;
    std::cout << "Number of particles: " << numParticles << std::endl;
    std::cout << "Use acceleration: " << useAcc << std::endl;

    // Check if the parsed data is valid
    if (sigma <= 0.0 || epsilon <= 0.0) {
    std::cerr << "Error: Invalid sigma or epsilon values. Exiting simulation." << std::endl;
    return -1;
}
// check if poisitions are valid 
if (positions_old.size() % 3 != 0) {
    std::cerr << "Error: Invalid number of position values. Exiting simulation." << std::endl;
    return -1;
}

    // the minimum x is 0.0
    // check if the positions are out of bounds
    if (boxSize > 0.000000001) { // to avoid numerical errors with very small box sizes
    for (const auto& pos : positions_old) {
        if (pos < 0.0 || pos > boxSize) {
            std::cerr << "Error: Positions are out of bounds!" << std::endl;
            return -1;
        }
    }
}

    //set box size to the maximum position + 0.5
    accelerations.resize(positions_old.size(), 0.0);
    forces.resize(positions_old.size(), 0.0);
    positions_new.resize(positions_old.size(), 0.0);
    velocities_new.resize(positions_old.size(), 0.0);
    
    //for each element calculate the forces
    for (size_t i = 0; i < positions_old.size(); i += 3) {
        force_updater(i, positions_old, forces, sigma, epsilon, boxSize);
        acceleration_calculator(i, forces, accelerations, masses);

    }

    //initialize the values on device
    double *positions_old_d, *velocities_old_d, *forces_d, *accelerations_d, *masses_d;
    double *positions_new_d, *velocities_new_d;
    int *cells_d;
    int *particleCell_d;
   
    // calculate the cell size and number of cells
    double cell_size;
    int num_cells;
    // if boxSize is 0.0 or cutoffRadius is 0.0, set num_cells to 1 and cell_size to boxSize
    // otherwise find the minimal divisor of cutoffRadius and boxSize
    if (boxSize == 0.0 || cutoffRadius == 0.0 || useAcc == 0) {
        num_cells = 1;
        cell_size = boxSize;
        std::cout << "Box size or cutoff radius or useAcc is zero, setting num_cells = 1, cell_size = " << cell_size << std::endl;
    } else {
        try {
            std::tie(cell_size, num_cells) = findMinimalDivisor(cutoffRadius, boxSize);
            std::cout << "Found cell_size: " << cell_size << ", with num_cells = " << num_cells << std::endl;
        } catch (const std::exception& e) {
            std::cerr << "Error: " << e.what() << std::endl;
        }
    }
    
    int total_cells = num_cells * num_cells * num_cells;

    // allocate memory on device
    CUDA_CHECK( hipMalloc(&positions_old_d, positions_old.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&velocities_old_d, velocities_old.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&forces_d, forces.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&accelerations_d, accelerations.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&masses_d, masses.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&positions_new_d, positions_new.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&velocities_new_d, velocities_new.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&particleCell_d, numParticles * sizeof(int)));
    CUDA_CHECK( hipMalloc(&cells_d, total_cells * sizeof(int)));

    // copy data to device
    CUDA_CHECK( hipMemcpy(positions_old_d, positions_old.data(), positions_old.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(velocities_old_d, velocities_old.data(), velocities_old.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(forces_d, forces.data(), forces.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(accelerations_d, accelerations.data(), accelerations.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(masses_d, masses.data(), masses.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(positions_new_d, positions_new.data(), positions_new.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(velocities_new_d, velocities_new.data(), velocities_new.size() * sizeof(double), hipMemcpyHostToDevice));

    // cout
    std::cout << "Data copied to device" << std::endl;
    

    // prepare the device kernel launch parameters
    dim3 blockSize(256);
    dim3 gridSize((masses.size() + blockSize.x - 1) / blockSize.x);
    // launch the kernel to check periodic boundaries


    //loop for GPU
    std::cout << "Starting time loop for GPU ..." << std::endl;
    // time loop
    auto start = std::chrono::high_resolution_clock::now();
    for (int timestep = 0; timestep < timeStepCount; ++timestep) {
        // cout
        //std::cout << "Time step: " << timestep << std::endl;
        //std::cout<< "updating positions and velocities"<< std::endl;
        update_positions_d<<<gridSize, blockSize>>>( positions_new_d, positions_old_d, 
            velocities_old_d, accelerations_d, timeStepLength, boxSize, numParticles);
        CUDA_CHECK(hipGetLastError());        
        update_velocities_d<<<gridSize, blockSize>>>(velocities_new_d, velocities_old_d,
             accelerations_d, timeStepLength, numParticles);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        //std::cout<< "Positions updated"<< std::endl;
        //std::cout<< "Update complete, swapping"<< std::endl;
        std::swap(positions_old_d, positions_new_d);
        std::swap(velocities_old_d, velocities_new_d);

        // Build up linked neighbor list
        // Reset cells to -1
        if (num_cells > 3) {
            // If there are only neighbor cells, we don't need to compute particle cells
            resetCells<<<total_cells, blockSize>>>(cells_d, total_cells); // should we lunch less blocks ? 
            computeParticleCells<<<gridSize, blockSize>>>(
                positions_old_d,
                cells_d,
                particleCell_d,
                numParticles,
                num_cells,
                total_cells,
                cell_size
            );
            
            /* debugging code to print particleCell_d and cells_d{
            // Copy particleCell_d and cells_d from device to host and print them
            std::vector<int> particleCell_host(numParticles);
            std::vector<int> cells_host(total_cells);

            CUDA_CHECK(hipMemcpy(particleCell_host.data(), particleCell_d, numParticles * sizeof(int), hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(cells_host.data(), cells_d, total_cells * sizeof(int), hipMemcpyDeviceToHost));
            
            // Print particleCell_d with timestep
            std::cout << "timestep " << timestep << " particleCell_d: ";
            for (int i = 0; i < numParticles; ++i) {
                std::cout << particleCell_host[i] << " ";
            }
            std::cout << std::endl;

            // Print cells_d with timestep
            std::cout << "timestep " << timestep << " cells_d: ";
            for (int i = 0; i < total_cells; ++i) {
                std::cout << cells_host[i] << " ";
            }
            std::cout << std::endl;
        }*/
        }
        
        //std::cout << " Calculating Forces and accelerations"<< std::endl;
        // update forces and accelerations
        acceleration_updater_d<<<gridSize, blockSize>>>(accelerations_d,positions_old_d, 
            forces_d, masses_d, sigma, epsilon, boxSize, cutoffRadius, numParticles, cells_d, particleCell_d, num_cells);
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipDeviceSynchronize());
        // cout
        //std::cout << "updating velocities"<< std::endl;
        // update velocities
        update_velocities_d<<<gridSize, blockSize>>>(velocities_new_d, velocities_old_d,
             accelerations_d, timeStepLength, numParticles);
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipDeviceSynchronize());
        // transfer new positions and velocities to old positions
        
        std::swap(velocities_old_d, velocities_new_d);
        //std::cout<< "Loop complete"<< std::endl;
        // print to file every printInterval steps
        if (printInterval > 0 && timestep % printInterval == 0) {
            // cout
            std::cout << "writing iteration " << timestep<<" to file"<< std::endl;
            hipMemcpy(positions_old.data(), positions_old_d, positions_old.size() * sizeof(double), hipMemcpyDeviceToHost);
            CUDA_CHECK(hipGetLastError());
            hipMemcpy(velocities_old.data(), velocities_old_d, velocities_old.size() * sizeof(double), hipMemcpyDeviceToHost);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
    
            std::string outputFile = "output/cuda-output" + std::to_string(timestep / printInterval) + ".vtk";
            writeVTKFile(outputFile, positions_old, velocities_old, masses);
        }
    }
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    std::cout << "Elapsed time on GPU: " << std::chrono::duration<double>(elapsed).count() << " seconds" << std::endl;
    
    std::cout << "Simulation complete!" << std::endl;

    // Free device memory
    hipFree(positions_old_d);
    hipFree(velocities_old_d);
    hipFree(forces_d);
    hipFree(accelerations_d);
    hipFree(masses_d);
    hipFree(positions_new_d);
    hipFree(velocities_new_d);
    hipFree(cells_d);
    hipFree(particleCell_d);
    
    
    return 0;
}