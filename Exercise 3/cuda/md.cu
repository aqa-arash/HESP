#include "hip/hip_runtime.h"
//write a quick test for the parser

#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <math.h>
#include <cmath>
#include "parser.hpp"
#include <chrono>
// cuda includes
#include "cudafuncs.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

// function to check and update periodic boundaries for each particle (can be globalized)
void checkPeriodicBoundaries(double & x, double & y, double & z, double boxSize) {
    x = fmod(fmod(x, boxSize) + boxSize, boxSize);
    y = fmod(fmod(y, boxSize) + boxSize, boxSize);
    z = fmod(fmod(z, boxSize) + boxSize, boxSize);
}


// function to calculate the periodic distance between two particles
// (can be globalized)
std::vector<double> periodic_distance(double x1, double y1, double z1, double x2, double y2, double z2, double boxSize) {
    std::vector<double> distances(3),pbc_distance(3);
    distances[0] = x1 - x2;
    distances[1] = y1 - y2;
    distances[2] = z1 - z2;
    // apply periodic boundary conditions
    pbc_distance[0] = distances[0] - boxSize * std::round(distances[0] / boxSize);
    pbc_distance[1] = distances[1] - boxSize * std::round(distances[1] / boxSize);
    pbc_distance[2] = distances[2] - boxSize * std::round(distances[2] / boxSize);
    return pbc_distance;
}

// function to calculate the distance size
// (can be globalized)
double distance_size(std::vector<double> & distances) {
    return std::sqrt(distances[0] * distances[0] + distances[1] * distances[1] + distances[2] * distances[2]);
}



// function to calculate the forces between two particles
// (can be globalized)
std::vector<double> ij_force_calculator(std::vector<double> distances, double sigma, double epsilon) {
    double r = distance_size(distances);
    if (r == 0.0) {
        std::cerr << "Error: Zero distance between particles!" << std::endl;
        return {0.0, 0.0, 0.0};
    }
    else if (r>2.5*sigma){ // cut off distance
        return {0.0, 0.0, 0.0};
    }
    else {
    double r6 = std::pow(sigma / r, 6);
    double force_multiplier = 24 * epsilon * r6 * (2 * r6 - 1) /(r*r);
    std::vector<double> forces(3);
    forces[0] = force_multiplier * distances[0];
    forces[1] = force_multiplier * distances[1];
    forces[2] = force_multiplier * distances[2];
    return forces;
}
}


// function to calculate the forces for a given particle
// (can be globalized)
void force_updater (size_t particle_idx, std::vector<double>& positions, std::vector<double>& forces, double sigma, double epsilon, double boxSize) {
    forces[particle_idx] = 0.0;
    forces[particle_idx + 1] = 0.0;
    forces[particle_idx + 2] = 0.0;
    // Calculate forces for the particle at particle_idx
    for (size_t j = 0; j < positions.size(); j += 3) {
        if (j != particle_idx) {
            std::vector<double> distances = periodic_distance(positions[particle_idx], positions[particle_idx + 1], positions[particle_idx + 2],
                positions[j], positions[j + 1], positions[j + 2], boxSize);
            std::vector<double> force = ij_force_calculator(distances, sigma, epsilon);
            forces[particle_idx] += force[0];
            forces[particle_idx + 1] += force[1];
            forces[particle_idx + 2] += force[2];
        }
    }    
}


// function to calculate the acceleration for a given particle
// (can be globalized)
void acceleration_calculator (int idx ,std::vector<double> & forces, std::vector<double> & acceleration, std::vector<double> mass ){
    acceleration[idx+0]= forces[idx+0]/mass[idx/3];
    acceleration[idx+1]= forces[idx+1]/mass[idx/3];
    acceleration[idx+2]= forces[idx+2]/mass[idx/3];
}




int main() {
    // Test file name
    std::string configFile = "config.txt";

    // Variables to hold parsed data
    std::vector<double> positions_old, velocities_old, masses, positions_new, velocities_new, accelerations, forces;
    
    double timeStepLength = 0.0, timeStepCount = 0.0, sigma = 0.0, epsilon = 0.0, boxSize = 0.0;
    int printInterval = 0;
    int numParticles = 0;

    
    // Call the parser
    parseConfigFile(configFile, positions_old, velocities_old, masses, boxSize, timeStepLength, timeStepCount, sigma, epsilon, printInterval);

    // Output the parsed data
    std::cout << "Parsed Data:" << std::endl;
    std::cout << "Time Step Length: " << timeStepLength << std::endl;
    std::cout << "Time Step Count: " << timeStepCount << std::endl;
    std::cout << "Sigma: " << sigma << std::endl;
    std::cout << "Epsilon: " << epsilon << std::endl;
    std::cout << "Box Size: " << boxSize << std::endl;
    std::cout << "Print Interval: " << printInterval << std::endl;  
    numParticles = positions_old.size()/3;
    std::cout << "Number of particles: " << numParticles << std::endl;
    
    // Check if the parsed data is valid
    if (sigma <= 0.0 || epsilon <= 0.0) {
    std::cerr << "Error: Invalid sigma or epsilon values. Exiting simulation." << std::endl;
    return -1;
}
    // the minimum x is 0.0
    // check if the positions are out of bounds
    for (const auto& pos : positions_old) {
        if (pos < 0.0 || pos > boxSize) {
            std::cerr << "Error: Positions are out of bounds!" << std::endl;
            return -1;
        }
    }

    //set box size to the maximum position + 0.5
    accelerations.resize(positions_old.size(), 0.0);
    forces.resize(positions_old.size(), 0.0);
    positions_new.resize(positions_old.size(), 0.0);
    velocities_new.resize(positions_old.size(), 0.0);
    
    //for each element calculate the forces
    for (size_t i = 0; i < positions_old.size(); i += 3) {
        force_updater(i, positions_old, forces, sigma, epsilon, boxSize);
        acceleration_calculator(i, forces, accelerations, masses);

    }

    //initialize the values on device
    double *positions_old_d, *velocities_old_d, *forces_d, *accelerations_d, *masses_d;
    double *positions_new_d, *velocities_new_d;
   

    // allocate memory on device
    CUDA_CHECK( hipMalloc(&positions_old_d, positions_old.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&velocities_old_d, velocities_old.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&forces_d, forces.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&accelerations_d, accelerations.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&masses_d, masses.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&positions_new_d, positions_new.size() * sizeof(double)));
    CUDA_CHECK( hipMalloc(&velocities_new_d, velocities_new.size() * sizeof(double)));


    // copy data to device
    CUDA_CHECK( hipMemcpy(positions_old_d, positions_old.data(), positions_old.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(velocities_old_d, velocities_old.data(), velocities_old.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(forces_d, forces.data(), forces.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(accelerations_d, accelerations.data(), accelerations.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(masses_d, masses.data(), masses.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(positions_new_d, positions_new.data(), positions_new.size() * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(velocities_new_d, velocities_new.data(), velocities_new.size() * sizeof(double), hipMemcpyHostToDevice));


    // cout
    std::cout << "Data copied to device" << std::endl;
    

    //hipMemcpy(positions_new.data(), positions_old_d, positions_old.size() * sizeof(double), hipMemcpyDeviceToHost);

    // prepare the device kernel launch parameters
    dim3 blockSize(256);
    dim3 gridSize((masses.size() + blockSize.x - 1) / blockSize.x);
    // launch the kernel to check periodic boundaries



    // write initial state to file
    std::string outputFile = "output/output0.vtk";
    writeVTKFile(outputFile, positions_old, velocities_old, masses);
    // cout
    //std::cout << "Initial state written to file " << outputFile << std::endl;
    // cout

    std::cout << "Starting time loop for CPU ..." << std::endl;
    // time loop
    auto start = std::chrono::high_resolution_clock::now();
    for (int timestep = 0; timestep < timeStepCount; ++timestep) {
        // cout
    //    std::cout << "Time step: " << timestep << std::endl;
        // cout
    //    std::cout << "updating positions and velocities"<< std::endl;
        // Update positions and velocities
        for (size_t i = 0; i < positions_old.size(); i += 3) {
            positions_new[i] = positions_old[i] + velocities_old[i] * timeStepLength + 0.5 * accelerations[i] * timeStepLength * timeStepLength;
            positions_new[i + 1] = positions_old[i + 1] + velocities_old[i + 1] * timeStepLength + 0.5 * accelerations[i + 1] * timeStepLength * timeStepLength;
            positions_new[i + 2] = positions_old[i + 2] + velocities_old[i + 2] * timeStepLength + 0.5 * accelerations[i + 2] * timeStepLength * timeStepLength;

            // check periodic boundaries
            checkPeriodicBoundaries(positions_new[i], positions_new[i + 1], positions_new[i + 2], boxSize);

            velocities_new[i] = velocities_old[i] + 0.5 * accelerations[i] * timeStepLength;
            velocities_new[i + 1] = velocities_old[i + 1] + 0.5 * accelerations[i + 1] * timeStepLength;
            velocities_new[i + 2] = velocities_old[i + 2] + 0.5 * accelerations[i + 2] * timeStepLength;
        }
        // cout
    //    std::cout << "Forces and accelerations"<< std::endl;
        // update forces and accelerations
        for (size_t i = 0; i < positions_old.size(); i += 3) {
            force_updater(i, positions_new, forces, sigma, epsilon, boxSize);
            acceleration_calculator(i, forces, accelerations, masses);
        }

        // transfer new velocities to old velocities
        std::swap(velocities_old, velocities_new);
        // cout
    //    std::cout << "updating velocities"<< std::endl;
        // update velocities
        for (size_t i = 0; i < positions_old.size(); i += 3) {
            velocities_new[i] = velocities_old[i] + 0.5 * accelerations[i]  * timeStepLength;
            velocities_new[i + 1] = velocities_old[i + 1] + 0.5 * accelerations[i + 1]  * timeStepLength;
            velocities_new[i + 2] = velocities_old[i + 2] + 0.5 * accelerations[i + 2]  * timeStepLength;
        }

        // transfer new positions and velocities to old positions
        std::swap(positions_old, positions_new);
        std::swap(velocities_old, velocities_new);

        // print to file every printInterval steps
        if (printInterval > 0 && timestep % printInterval == 0) {
            // cout
            std::cout << "writing iteration " << timestep<<" to file"<< std::endl;
        
            std::string outputFile = "output/output" + std::to_string(timestep / printInterval) + ".vtk";
            writeVTKFile(outputFile, positions_old, velocities_old, masses);
        }
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Elapsed time on cpu: " << elapsed.count() << " seconds" << std::endl;


    //loop for GPU
    std::cout << "Starting time loop for GPU ..." << std::endl;
    // time loop
    start = std::chrono::high_resolution_clock::now();
    for (int timestep = 0; timestep < timeStepCount; ++timestep) {
        // cout
        //std::cout << "Time step: " << timestep << std::endl;
        //std::cout<< "updating positions and velocities"<< std::endl;
        update_positions_d<<<gridSize, blockSize>>>( positions_new_d, positions_old_d, 
            velocities_old_d, accelerations_d, timeStepLength, boxSize, numParticles);
        CUDA_CHECK(hipGetLastError());
        //std::cout<< "Positions updated"<< std::endl;
        
        update_velocities_d<<<gridSize, blockSize>>>(velocities_new_d, velocities_old_d,
             accelerations_d, timeStepLength, numParticles);
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipDeviceSynchronize());

        //std::cout<< "Update complete, swapping"<< std::endl;
        std::swap(positions_old_d, positions_new_d);
        std::swap(velocities_old_d, velocities_new_d);
        // cout
        //std::cout << " Calculating Forces and accelerations"<< std::endl;
        // update forces and accelerations
        acceleration_updater_d<<<gridSize, blockSize>>>(accelerations_d,positions_old_d, 
            forces_d, masses_d, sigma, epsilon, boxSize, numParticles);
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipDeviceSynchronize());
        // cout
        //std::cout << "updating velocities"<< std::endl;
        // update velocities
        update_velocities_d<<<gridSize, blockSize>>>(velocities_new_d, velocities_old_d,
             accelerations_d, timeStepLength, numParticles);
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipDeviceSynchronize());
        // transfer new positions and velocities to old positions
        
        std::swap(velocities_old_d, velocities_new_d);
        //std::cout<< "Loop complete"<< std::endl;
        // print to file every printInterval steps
        if (printInterval > 0 && timestep % printInterval == 0) {
            // cout
            std::cout << "writing iteration " << timestep<<" to file"<< std::endl;
            hipMemcpy(positions_old.data(), positions_old_d, positions_old.size() * sizeof(double), hipMemcpyDeviceToHost);
            CUDA_CHECK(hipGetLastError());
            hipMemcpy(velocities_old.data(), velocities_old_d, velocities_old.size() * sizeof(double), hipMemcpyDeviceToHost);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
            std::string outputFile = "output/cuda-output" + std::to_string(timestep / printInterval) + ".vtk";
            writeVTKFile(outputFile, positions_old, velocities_old, masses);
        }
    }
    end = std::chrono::high_resolution_clock::now();
    elapsed = end - start;
    std::cout << "Elapsed time on GPU: " << elapsed.count() << " seconds" << std::endl;
    
    std::cout << "Simulation complete!" << std::endl;

    // Free device memory
    hipFree(positions_old_d);
    hipFree(velocities_old_d);
    hipFree(forces_d);
    hipFree(accelerations_d);
    hipFree(masses_d);
    hipFree(positions_new_d);
    hipFree(velocities_new_d);
    
    
    return 0;
}